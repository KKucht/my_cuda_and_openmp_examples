#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <omp.h>
#include <time.h>

using namespace cv;

#define N 10

__constant__ int Gx[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int Gy[3][3] = {
    {-1, -2, -1},
    { 0,  0,  0},
    { 1,  2,  1}
};

__global__ void sobel_operator(unsigned char* in_image, unsigned char* out_image, long long width, long long height) {
    long long int x = blockIdx.x * blockDim.x + threadIdx.x;
    long long int y = blockIdx.y * blockDim.y + threadIdx.y;

    long long local_x = x + 1;
    long long local_y = y + 1;

    if (local_x < width - 1 && local_y < height - 1) {
            
        int sumx = 0;
        int sumy = 0;
        for (int p = -1; p <= 1; p++) {
            for (int q = -1; q <= 1; q++) {
                long long idx = (local_y + p) * width + (local_x + q);
                sumx += (in_image[idx] * Gx[p + 1][q + 1]);
                sumy += (in_image[idx] * Gy[p + 1][q + 1]);
            }
        }

        int magnitude = sqrtf(sumx * sumx + sumy * sumy);

        long long idx_out = local_y * width + local_x;
        out_image[idx_out] = (unsigned char)(magnitude > 255 ? 255 : magnitude);

    }
}

__global__ void sobel_operator_empty(unsigned char* in_image, unsigned char* out_image, long long width, long long height) {
}

void checkCudaError(const char *message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

void my_main1(){


    cv::Mat img = cv::imread("img.jpg", CV_8UC1);
    
    unsigned char *image_array1;
    unsigned char *image_array2;

    long long rows = img.rows;
    long long cols = img.cols;
    long long size = 0;

    printf("rows: %lld\ncols: %lld\n", rows, cols);
    size = rows * cols * sizeof(unsigned char);
    image_array2 = (unsigned char *)malloc(size);
    image_array1 = (unsigned char *)malloc(size);

    if (image_array1 == NULL || image_array2 == NULL) {
        printf("Nie udało się zaalokować pamięci.\n");
        return;
    }

    for (unsigned long long int i = 0; i < rows; ++i) {
        memcpy(image_array1 + i * cols, img.ptr(i), cols * sizeof(unsigned char));
    }
    
    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);
    checkCudaError("Memory allocation");

    hipMemcpy(d_input, image_array1, size, hipMemcpyHostToDevice);
    checkCudaError("Memory copy to device");

    dim3 threadsPerBlock(32, 32);

    dim3 blocksPerGrid(
        (cols + threadsPerBlock.x - 1 ) / threadsPerBlock.x,
        (rows + threadsPerBlock.y - 1 ) / threadsPerBlock.y
    );
    

    sobel_operator<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, cols, rows);
    hipDeviceSynchronize();
    checkCudaError("Kernel execution");

    hipMemcpy(image_array2, d_output, size, hipMemcpyDeviceToHost);

    printf("Generate image\n");
    cv::Mat new_image(rows, cols, CV_8UC1);
    for (unsigned long long int i = 0; i < rows; ++i) {
        memcpy(new_image.ptr(i), image_array2 + i * cols, cols * sizeof(unsigned char));
    }

    cv::imwrite("imgout.png", new_image);

    
    
    hipFree(d_input);
    hipFree(d_output);
    free(image_array1);
    free(image_array2);
}

void my_main2(){
    cv::Mat img = cv::imread("img.jpg", CV_8UC1);
    
    unsigned char *image_array1;
    unsigned char *image_array2;

    long long rows = img.rows;
    long long cols = img.cols;
    long long size = 0;

    printf("rows: %lld\ncols: %lld\n", rows, cols);
    size = rows * cols * sizeof(unsigned char);
    image_array2 = (unsigned char *)malloc(size);
    image_array1 = (unsigned char *)malloc(size);

    if (image_array1 == NULL || image_array2 == NULL) {
        printf("Nie udało się zaalokować pamięci.\n");
        return;
    }

    #pragma omp parallel for
    for (unsigned long long int i = 0; i < rows; ++i) {
        memcpy(image_array1 + i * cols, img.ptr(i), cols * sizeof(unsigned char));
    }
    
    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);
    checkCudaError("Memory allocation");

    hipMemcpy(d_input, image_array1, size, hipMemcpyHostToDevice);
    checkCudaError("Memory copy to device");

    dim3 threadsPerBlock(32, 32);

    dim3 blocksPerGrid(
        (cols + threadsPerBlock.x - 1 ) / threadsPerBlock.x,
        (rows + threadsPerBlock.y - 1 ) / threadsPerBlock.y
    );

    sobel_operator<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, cols, rows);
    hipDeviceSynchronize();
    checkCudaError("Kernel execution");

    hipMemcpy(image_array2, d_output, size, hipMemcpyDeviceToHost);

    printf("Generate image\n");
    cv::Mat new_image(rows, cols, CV_8UC1);

    #pragma omp parallel for
    for (unsigned long long int i = 0; i < rows; ++i) {
        memcpy(new_image.ptr(i), image_array2 + i * cols, cols * sizeof(unsigned char));
    }

    cv::imwrite("imgout.png", new_image);
    
    hipFree(d_input);
    hipFree(d_output);
    free(image_array1);
    free(image_array2);

}

int main(int argc, char **argv) {
    double start_time = omp_get_wtime();
    my_main1();
    double end_time = omp_get_wtime();
    printf("Time taken: %f seconds\n", end_time - start_time);
    start_time = omp_get_wtime();
    my_main2();
    end_time = omp_get_wtime();
    printf("Time taken: %f seconds\n", end_time - start_time);
    
    return 0;
}
