/*
CUDA - generation and sum of arithmetic progression build of 10240000 elements a1=0 r=1 with Unified Memory
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

//elements generation
__global__ 
void calculate(int *result) {
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    result[my_index]=my_index;
}


int main(int argc,char **argv) {

    long long result;
    int threadsinblock=1024;
    int blocksingrid=10000;	

    int size = threadsinblock*blocksingrid;

    int *results;

    //unified memory allocation - available for host and device
    if (hipSuccess!=hipMallocManaged(&results,size*sizeof(int)))
      errorexit("Error allocating memory on the GPU");

    //call to GPU - kernel execution 
    calculate<<<blocksingrid,threadsinblock>>>(results);

    if (hipSuccess!=hipGetLastError())
      errorexit("Error during kernel launch");
  
    //device synchronization to ensure that data in memory is ready
    hipDeviceSynchronize();

    //calculate sum of all elements
    result=0;
    for(int i=0;i<size;i++) {
      result = result + results[i];
    }

    printf("\nSum of all elements is  %lld\n",result);

    //free memory
    if (hipSuccess!=hipFree(results))
      errorexit("Error when deallocating space on the GPU");

}
