#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <time.h>
#include "raw_image_reader.hpp"

using namespace cv;

#define N 10

void read_data(unsigned char* in_image, unsigned char* out_image, long long int width, long long int height, long long int padded_width) {
    for (long long int i = 0; i < height ; i++) {
        memcpy(out_image + i * width, in_image + (i + 1) * padded_width + 1, width * sizeof(unsigned char));
    }
}

void prep_data(unsigned char* in_image, unsigned char* out_image, long long int width, long long int height, long long int padded_width) {
    for (long long int i = 0; i < height ; ++i) {
        memcpy(out_image + (i + 1) * padded_width + 1, in_image + i * width, width * sizeof(unsigned char));
    }
}

__global__ void sobel_operator(unsigned char* in_image, unsigned char* out_image, long long width, long long height) {
    long long int x = blockIdx.x * blockDim.x + threadIdx.x;
    long long int y = blockIdx.y * blockDim.y + threadIdx.y;

    long long local_x = x + 1;
    long long local_y = y + 1;

    int sumx = 0;
    int sumy = 0;

    sumx -=      in_image[(local_y - 1)* width + local_x  - 1];
    sumx +=      in_image[(local_y - 1)* width + local_x  + 1];
    sumx -=  2 * in_image[(local_y    )* width + local_x  - 1];
    sumx +=  2 * in_image[(local_y    )* width + local_x  + 1];
    sumx -=      in_image[(local_y + 1)* width + local_x  - 1];
    sumx +=      in_image[(local_y + 1)* width + local_x  + 1];

    sumy -=      in_image[(local_y - 1)* width + local_x  - 1];
    sumy -=  2 * in_image[(local_y - 1)* width + local_x     ];
    sumy -=      in_image[(local_y - 1)* width + local_x  + 1];
    sumy +=      in_image[(local_y + 1)* width + local_x  - 1];
    sumy +=  2 * in_image[(local_y + 1)* width + local_x     ];
    sumy +=      in_image[(local_y + 1)* width + local_x  + 1];

    int magnitude = sqrtf(sumx * sumx + sumy * sumy);

    long long idx_out = local_y * width + local_x;
    out_image[idx_out] = (unsigned char)(magnitude > 255 ? 255 : magnitude);
}

__global__ void sobel_operator_empty(unsigned char* in_image, unsigned char* out_image, long long width, long long height) {
}

void checkCudaError(const char *message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

int main(int argc, char **argv) {
    
    unsigned char *image_array1;
    unsigned char *image_array2;

    long long rows = 0;
    long long cols = 0;
    long long size = 0;
    double max = 0, min=0;

    if (!raw::readImageRAW("imgin", image_array1, cols, rows)) {
        printf("Nie można otworzyć lub znaleźć obrazu.\n");
        return -1;
    }

    printf("rows: %lld\ncols: %lld\n", rows, cols);
    size = rows * cols * sizeof(unsigned char);
    image_array2 = (unsigned char *)malloc(size);

    if (image_array1 == NULL || image_array2 == NULL) {
        printf("Nie udało się zaalokować pamięci.\n");
        return -1;
    }

    printf("There will be avarege time for N = %d.\n", N);

    dim3 threadsPerBlock(32, 32);

    /// PREPERE, to NEW BLOCKS ADDITIONAL
    long long int real_nr_of_blocks_x = (cols + threadsPerBlock.x - 1) / (threadsPerBlock.x);
    long long int real_nr_of_blocks_y = (rows + threadsPerBlock.y - 1) / (threadsPerBlock.y);

    long long int padded_width = real_nr_of_blocks_x * (threadsPerBlock.x) + 2;
    long long int padded_height = real_nr_of_blocks_y * (threadsPerBlock.y) + 2;
    size = padded_width * padded_height ;

    unsigned char * new_in_image = (unsigned char *)calloc(size , sizeof(unsigned char));
    unsigned char * new_out_image = (unsigned char *)calloc(size , sizeof(unsigned char));

    printf("new rows: %lld\nnew cols: %lld\n", padded_height, padded_width);

    size *= sizeof(unsigned char);

    prep_data (image_array1, new_in_image, cols, rows, padded_width);

    dim3 blocksPerGrid(
        real_nr_of_blocks_x,
        real_nr_of_blocks_y
    );

    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);
    checkCudaError("Memory allocation");

    hipMemcpy(d_input, image_array1, size, hipMemcpyHostToDevice);
    checkCudaError("Memory copy to device");
    // init clocks
    double elapsed_time;
    clock_t start_time1, end_time1, start_time2, end_time2;
    clock_t start_timers1[N], end_timers1[N], start_timers2[N], end_timers2[N];

    

    start_time1 = clock();
    for (int i = 0 ; i< N ; i++ ) {
        start_timers1[i] = clock();
        sobel_operator<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, padded_width, padded_height);
        hipDeviceSynchronize();
        checkCudaError("Kernel execution");
        end_timers1[i] = clock();
    }
    end_time1 = clock();

    start_time2 = clock();
    for (int i = 0 ; i< N ; i++ ) {
        start_timers2[i] = clock();
        sobel_operator_empty<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, cols, rows);
        hipDeviceSynchronize();
        checkCudaError("Kernel execution");
        end_timers2[i] = clock();
    }
    end_time2 = clock();

    elapsed_time = (double)((end_time1 - start_time1) - (end_time2 - start_time2)) / (CLOCKS_PER_SEC/1000) / N;
    printf("Second: Sobel operator completed in %.10f miliseconds\n", elapsed_time);

    max = -99;
    min = 9999999;
    for (int i = 0; i < N; i++){
        elapsed_time = (double)((end_timers1[i] - start_timers1[i]) - (end_timers2[i] - start_timers2[i])) / (CLOCKS_PER_SEC/1000) / N;
        if (min > elapsed_time){
            min = elapsed_time;
        }
        if (max < elapsed_time){
            max = elapsed_time;
        }
    }

    printf("Uncertainty: %.10f\n",  (max - min) / 2.0);

    hipMemcpy(new_out_image, d_output, size, hipMemcpyDeviceToHost);
    checkCudaError("Memory copy to host");

    read_data(new_out_image, image_array2, cols, rows, padded_width);

    free (new_out_image);


    printf("Generate image\n");
    cv::Mat new_image(rows, cols, CV_8UC1);
    for (unsigned long long int i = 0; i < rows; ++i) {
        memcpy(new_image.ptr(i), image_array2 + i * cols, cols * sizeof(unsigned char));
    }

    cv::imwrite("imgout.png", new_image);
    
    hipFree(d_input);
    hipFree(d_output);
    free(image_array1);
    free(image_array2);

    return 0;
}
